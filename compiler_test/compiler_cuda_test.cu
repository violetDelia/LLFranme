#include "compiler_cuda_test.cuh"
#include "hip/hip_runtime.h"
#include ""


float * free_to_host(float *host,float *gpu,size_t length,int device){
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    hipMemcpy((void*)host,(void*)gpu,length,hipMemcpyDeviceToHost);
    hipFree(gpu);
    return host;
}

float *malloc_cuda(float *host,size_t length,int device){
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    float *GPU;
    hipMalloc((void**)&GPU,length);
    hipMemcpy((void*)GPU,(void*)host,length,hipMemcpyHostToDevice);
    return GPU;
}


void add_one(float *Gpu,int length,int device){
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    int threadMaxSize = devProp.maxThreadsPerBlock;
    int blockSize = ((length/sizeof(float))+threadMaxSize-1)/threadMaxSize;
    dim3 thread(threadMaxSize);
    dim3 block(blockSize);
    add_one_cuda<float><<<block,thread>>>(Gpu);
}

float *matAdd(float *a,float *b,int length)
{
    int device = 0;//设置使用第0块GPU进行运算
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    int threadMaxSize = devProp.maxThreadsPerBlock;
    int blockSize = (length+threadMaxSize-1)/threadMaxSize;
    dim3 thread(threadMaxSize);
    dim3 block(blockSize);
    int size = length * sizeof(float);
    float *sum =(float *)malloc(size) ;
    float *sumGPU,*aGPU,*bGPU;
    hipMalloc((void**)&sumGPU,size);
    hipMalloc((void**)&aGPU,size);
    hipMalloc((void**)&bGPU,size);
    //内存->显存
    hipMemcpy((void*)aGPU,(void*)a,size,hipMemcpyHostToDevice);
    hipMemcpy((void*)bGPU,(void*)b,size,hipMemcpyHostToDevice);
    //运算
    matAdd_cuda<float><<<block,thread>>>(aGPU,bGPU,sumGPU);
    //hipDeviceSynchronize();
    //显存->内存
    hipMemcpy(sum,sumGPU,size,hipMemcpyDeviceToHost);
    //释放显存
    hipFree(sumGPU);
    hipFree(aGPU);
    hipFree(bGPU);
    return sum;
}

