#include "compiler_cuda_test.cuh"
#include "hip/hip_runtime.h"
#include ""

float *matAdd(float *a,float *b,int length)
{
    int device = 0;//设置使用第0块GPU进行运算
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    int threadMaxSize = devProp.maxThreadsPerBlock;
    int blockSize = (length+threadMaxSize-1)/threadMaxSize;
    dim3 thread(threadMaxSize);
    dim3 block(blockSize);
    int size = length * sizeof(float);
    float *sum =(float *)malloc(size) ;
    float *sumGPU,*aGPU,*bGPU;
    hipMalloc((void**)&sumGPU,size);
    hipMalloc((void**)&aGPU,size);
    hipMalloc((void**)&bGPU,size);
    //内存->显存
    hipMemcpy((void*)aGPU,(void*)a,size,hipMemcpyHostToDevice);
    hipMemcpy((void*)bGPU,(void*)b,size,hipMemcpyHostToDevice);
    //运算
    matAdd_cuda<float><<<block,thread>>>(aGPU,bGPU,sumGPU);
    //hipDeviceSynchronize();
    //显存->内存
    hipMemcpy(sum,sumGPU,size,hipMemcpyDeviceToHost);
    //释放显存
    hipFree(sumGPU);
    hipFree(aGPU);
    hipFree(bGPU);
    return sum;
}